
#include <hip/hip_runtime.h>
#include<cstdio>
#include "hip/hip_vector_types.h"

extern "C" {


__global__ void cuAdd(int* list, int* elements, int i, int listSize){
	int thid = blockIdx.x * blockDim.x + threadIdx.x;
	while(thid < listSize){
		int value = list[thid] + elements[i];
		list[thid+listSize] = value;
		thid += blockDim.x * gridDim.x;
	}
}

__device__ int2 findMedian(int* tabA, int a, int b, int* tabB, int c, int d);

__global__ void cuPartition(int j, int* prevList, int4* H, int size){
	int* newList = prevList+size;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int threadCounts = 1 << (j-1);
	while (tid < threadCounts){
		int medianId = tid + threadCounts;
		int a = H[medianId].x;
		int b = H[medianId].y;
		int c = H[medianId].z;
		int d = H[medianId].w;
		int2 ef = findMedian(prevList, a, b, newList, c, d);
		H[2*medianId].x = a;
		H[2*medianId].y = ef.x;
		H[2*medianId].z = c;
		H[2*medianId].w = ef.y;
		H[2*medianId + 1].x = ef.x;
		H[2*medianId + 1].y = b;
		H[2*medianId + 1].z = ef.y;
		H[2*medianId + 1].w = d;
		tid += blockDim.x * gridDim.x;
	}
}

__device__ void mergeInc(int* listA, int beginA, int endA, int* listB, int beginB, int endB, int* result);

__global__ void cuMergeIncreasing(int* lists, int4* H, int listSize, int threads, int* result){
	int* newList = lists + listSize;
	int tid = blockIdx.x * blockDim.x + threadIdx.x + 1;
	while(tid <= threads){
		int medianId = tid + threads - 1;
		int4 localFetch = H[medianId];
		int a = localFetch.x;
		int b = localFetch.y;
		int c = localFetch.z;
		int d = localFetch.w;
		mergeInc(lists, a, b, newList, c, d, result);
		tid += blockDim.x * gridDim.x;
	}
}

__global__ void cuPrune(int* listA, int sizeA, int* listB, int sizeB, int* found, int2* pickedBlocks, int* pickedBlocksCounter, int M){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int k = blockDim.x*gridDim.x;
	int chunkA = (sizeA + k - 1)/ (k);
	int chunkB = (sizeB + k - 1)/ (k);
	for(int j = 0; j < k ; j++){
		if(*found) return;
		int x = listA[tid * chunkA] + listB[(j+1) * chunkB - 1]; // mozemy wyskoczyc jesli chunkA lub ChunbB nie dzieli k
		int y = listA[(tid+1) * chunkA - 1] + listB[j * chunkB]; // mozemy wyskoczyc tez
		if (x == M || y == M) atomicExch(found, 1);
		else if(x < M && y > M){
			int pos = atomicAdd(pickedBlocksCounter, 1);
			pickedBlocks[pos].x = tid;
			pickedBlocks[pos].y = j;
		}
	}
}

__device__ bool searchSteep(int* listA, int chunkSizeA, int* listB, int chunkSizeB, int M){
	int a, b;
	a = b = 0;
	while(a < chunkSizeA && b < chunkSizeB){
		int value = listA[a] + listB[b];
		if(value == M) return true;
		if(value < M) a++;
		else b++;
	}
	return false;
}

__global__ void cuSearch(int* listA, int sizeA, int* listB, int sizeB, int2* pickedBlocks, int* noPickedBlocks, int* found, int M){
	int thid = blockIdx.x * blockDim.x + threadIdx.x;
	int k = blockDim.x*gridDim.x;
	int chunkA = (sizeA + k - 1)/ (k);
	int chunkB = (sizeB + k - 1)/ (k);
	while(thid < *noPickedBlocks){
		if(*found) return;
		int2 idsOfFragmentToCheck = pickedBlocks[thid];
		int* shiftedListA = listA + idsOfFragmentToCheck.x * chunkA;
		int* shiftedListB = listB + idsOfFragmentToCheck.y * chunkB;
		int _sizeA = thid != k-1 ? chunkA : sizeA % chunkA;
		int _sizeB = thid != k-1 ? chunkB : sizeB % chunkB;

		bool f = searchSteep(shiftedListA, _sizeA, shiftedListB, _sizeB, M);
		if(f) *found = true;

		thid += k;
	}
}

__global__ void cuReverse(int* tab, int size){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid >= size/2)
		return;
	int tmp = tab[tid];
	tab[tid] = tab[size-tid-1];
	tab[size-tid-1] = tmp;
}

__device__ int binsearchInc(int* tab, int l, int r, int value){
	while(l < r){
		int m = (l + r) / 2;
		if(tab[m] >= value){
			r = m;
		} else{
			l = m+1;
		}
	}
	return l;
}


__device__ int2 findMedian(int* tabA, int a, int b, int* tabB, int c, int d){
	int aMiddle, bMiddle, otherBegin, otherEnd, otherValue;
	int* otherTab;
	if(b-a > d-c){
		aMiddle = (b + a) / 2;
		otherTab = tabB;
		otherBegin = c;
		otherEnd = d;
		otherValue = tabA[aMiddle];
		//bMiddle = binsearchInc(tabB, c, d, tabA[aMiddle]);
	} else{
		bMiddle = (c + d) / 2;
		otherTab = tabA;
		otherBegin = a;
		otherEnd = b;
		otherValue = tabB[bMiddle];
		//aMiddle = binsearchInc(tabA, a, b, tabB[bMiddle]);
	}
	int theOtherMiddle = binsearchInc(otherTab, otherBegin, otherEnd, otherValue);
	if(b-a > d-c){
		bMiddle = theOtherMiddle;
	} else{
		aMiddle = theOtherMiddle;
	}
	int2 result;
	result.x = aMiddle;
	result.y = bMiddle;
	return result;
}

__device__ inline void mergeInc(int* listA, int beginA, int endA, int* listB, int beginB, int endB, int* result){
	int position = beginA + beginB;
	while(beginA < endA && beginB < endB){
		if (listA[beginA] < listB[beginB]){
			result[position++] = listA[beginA++];
		} else{
			result[position++] = listB[beginB++];
		}
	}

	while(beginA < endA){
		result[position++] = listA[beginA++];
	}
	while(beginB < endB){
		result[position++] = listB[beginB++];
	}

}



}
